#include "hip/hip_runtime.h"
/*
 * Copyright © Advanced Micro Devices, Inc. All rights reserved.
 * Adapted from
 * https://github.com/sgl-project/sglang/blob/main/sgl-kernel/csrc/moe/moe_fused_gate.cu
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include <torch/all.h>

#include <cfloat>
#include <type_traits>

#include "hip_compat.h"
#include "hip_reduce.h"
#include "vec_convert.h"
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>

/// Aligned array type
template <typename T,
          /// Number of elements in the array
          int N,
          /// Alignment requirement in bytes
          int Alignment = sizeof(T) * N>
class alignas(Alignment) AlignedArray
{
    float data[N];
};

using bfloat16_t = ck_tile::bfloat16_t;
using float16_t  = ck_tile::half_t;
using float32_t  = float;

// QQ NOTE: to handle the case for at::Half, error: more than one operator ">" matches these
// operands: built-in operator "arithmetic > arithmetic" function "operator>(const __half &, const
// __half &)"
template <typename T>
__device__ inline bool cmp_gt(const T& a, const T& b)
{
    if constexpr(std::is_same<T, float16_t>::value || std::is_same<T, bfloat16_t>::value)
    {
        // at::Half (or float16_t in our native case) causes ambiguity, so we cast to float.
        return ck_tile::type_convert<float>(a) > ck_tile::type_convert<float>(b);
    }
    else
    {
        // For types like float, at::BFloat16, or cutlass::half_t / cutlass::bfloat16_t, assume
        // operator> works as expected.
        return a > b;
    }
}

template <typename T>
__device__ inline bool cmp_eq(const T& a, const T& b)
{
    if constexpr(std::is_same<T, float16_t>::value || std::is_same<T, bfloat16_t>::value)
    {
        return ck_tile::type_convert<float>(a) == ck_tile::type_convert<float>(b);
    }
    else
    {
        return a == b;
    }
}

// Fixed constants common to both dynamic and static template versions:
// static constexpr int WARP_SIZE = 32;
static constexpr int WARPS_PER_CTA = 1;
static constexpr int MAX_VPT =
    32; // maximum VPT we support, > params.VPT = num_expert / num_expert_group

// Create an alias for Array using AlignedArray
// template <typename T, int N>
// using Array = AlignedArray<T, N>;
// QQ: NOTE expression must have a constant value, this has to be > params.VPT
// template <typename T>
// using AccessType = AlignedArray<T, MAX_VPT>;

template <typename T, typename Params>
__device__ void moe_fused_gate_impl(void* input,
                                    void* bias,
                                    float* output_ptr,
                                    int32_t* indices_ptr,
                                    int64_t num_rows,
                                    int64_t topk_group,
                                    int64_t topk,
                                    int64_t num_fused_shared_experts,
                                    double routed_scaling_factor,
                                    Params params)
{
    int tidx           = threadIdx.x;
    int64_t thread_row = blockIdx.x * params.ROWS_PER_CTA + threadIdx.y * params.ROWS_PER_WARP +
                         tidx / params.THREADS_PER_ROW;
    if(thread_row >= num_rows)
    {
        return;
    }
    extern __shared__ char shared_mem[];
    char* ptr = (char*)(((size_t)shared_mem + 255) & ~255);

    // float *scores = reinterpret_cast<float *>(ptr + tidx / params.THREADS_PER_ROW *
    // params.THREADS_PER_ROW * params.VPT * sizeof(float)); ptr += WARP_SIZE * params.VPT *
    // sizeof(float);

    float* scores =
        reinterpret_cast<float*>(ptr + tidx / params.THREADS_PER_ROW * topk * sizeof(float));
    ptr += params.ROWS_PER_WARP * topk * sizeof(float);

    int* topk_indices =
        reinterpret_cast<int*>(ptr + tidx / params.THREADS_PER_ROW * topk * sizeof(int));
    // ptr += params.ROWS_PER_WARP * topk * sizeof(int);

    // Calculate topk_excluding_share_expert_fusion from topk
    int64_t topk_excluding_share_expert_fusion = topk - num_fused_shared_experts;

    // Cast pointers to type T:
    auto* input_ptr      = reinterpret_cast<T*>(input);
    auto* bias_ptr       = reinterpret_cast<T*>(bias);
    auto* thread_row_ptr = input_ptr + thread_row * params.NUM_EXPERTS;

    int thread_group_idx         = tidx % params.THREADS_PER_ROW;
    int first_elt_read_by_thread = thread_group_idx * params.VPT;

    // Create local arrays for the row chunk and bias chunk and then reinterpret the address of
    // row_chunk as a pointer to AccessType.

    // constexpr uint32_t vec_size = 16 / sizeof(T);
    using AccessType = ck_tile::vec_t<T, MAX_VPT>;
    using VecType    = ck_tile::vec_t<float, MAX_VPT>;

    T* thread_read_ptr = thread_row_ptr + first_elt_read_by_thread;
    VecType row_chunk;
    AccessType const* vec_thread_read_ptr = reinterpret_cast<AccessType const*>(thread_read_ptr);

    T* bias_thread_read_ptr = bias_ptr + first_elt_read_by_thread;
    VecType bias_chunk;
    AccessType const* vec_bias_thread_read_ptr =
        reinterpret_cast<AccessType const*>(bias_thread_read_ptr);

    // QQ NOTE: doing the follow will be slower than loop assign and more importantly
    // have misaligned address issue when params.VPT < 8 and mismatch with MAX_VPT
    // AccessType<T>* row_chunk_vec_ptr = reinterpret_cast<AccessType<T>*>(&row_chunk);
    // row_chunk_vec_ptr[0] = vec_thread_read_ptr[0];
    // bias_chunk_vec_ptr[0] = vec_bias_thread_read_ptr[0];
    // #pragma unroll
    //   for (int ii = 0; ii < params.VPT; ++ii) {
    //     row_chunk_vec_ptr[ii] = vec_thread_read_ptr[0][ii];
    //     bias_chunk_vec_ptr[ii] = vec_bias_thread_read_ptr[0][ii];
    //   }]

    AccessType row_chunk_vec        = *vec_thread_read_ptr;
    AccessType bias_thread_read_vec = *vec_bias_thread_read_ptr;
    for(int jj = 0; jj < params.VPT; ++jj)
    {
        row_chunk[jj]  = ck_tile::type_convert<float>(row_chunk_vec(jj));
        bias_chunk[jj] = ck_tile::type_convert<float>(bias_thread_read_vec(jj));
    }
    // #pragma unroll
    // for (int ii = 0; ii < params.VPT / vec_size; ++ii) {
    //   AccessType row_chunk_vec = vec_thread_read_ptr[ii];
    //   AccessType bias_thread_read_vec = vec_bias_thread_read_ptr[ii];
    //   for (int jj = 0; jj < vec_size; ++jj) {
    //     row_chunk[ii * vec_size + jj] = ck_tile::type_convert<float>(row_chunk_vec(jj));
    //     bias_chunk[ii * vec_size + jj] = ck_tile::type_convert<float>(bias_thread_read_vec(jj));
    //   }
    // }

    // __syncthreads();

////////////////////// Sigmoid //////////////////////
#pragma unroll
    for(int ii = 0; ii < params.VPT; ++ii)
    {
        row_chunk[ii] = 1.0f / (1.0f + expf(-row_chunk[ii]));
    }
    // __syncthreads();

////////////////////// Add Bias //////////////////////
#pragma unroll
    for(int ii = 0; ii < params.VPT; ++ii)
    {
        bias_chunk[ii] = row_chunk[ii] + bias_chunk[ii];
    }

    // local argmax
    float max_val        = -FLT_MAX;
    float max_val_second = -FLT_MAX;
#pragma unroll
    for(int ii = 0; ii < params.VPT; ++ii)
    {
        float val = bias_chunk[ii];

        if(cmp_gt(val, max_val))
        {
            max_val_second = max_val;
            max_val        = val;
        }
        else if(cmp_gt(val, max_val_second))
        {
            max_val_second = val;
        }
    }
    // QQ NOTE: currently fixed to pick top2 sigmoid weight value in each expert group and sum them
    // as the group weight to select expert groups
    max_val = max_val + max_val_second;

////////////////////// Exclude Groups //////////////////////
#pragma unroll
    for(int k_idx = 0; k_idx < params.THREADS_PER_ROW - topk_group; ++k_idx)
    { // QQ NOTE Here params.THREADS_PER_ROW = num_expert_group
        int expert    = first_elt_read_by_thread;
        float max_sum = max_val;

        // // argmin reduce
        // #pragma unroll
        //     for (int mask = params.THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
        //       float other_max_sum =
        //           VLLM_SHFL_XOR_SYNC_WIDTH(max_sum, mask, params.THREADS_PER_ROW);
        //       int other_expert = VLLM_SHFL_XOR_SYNC_WIDTH(expert, mask, params.THREADS_PER_ROW);

        //       // higher indices win
        //       if (cmp_gt(max_sum, other_max_sum) || (cmp_eq(other_max_sum, max_sum) &&
        //       other_expert > expert)) {
        //         max_sum = other_max_sum;
        //         expert = other_expert;
        //       }
        //     }

        using kvp = hipcub::KeyValuePair<int, float>;

        hipcub::ArgMax arg_max;
        hipcub::ArgMin arg_min;

        kvp thread_kvp;
        thread_kvp.key       = expert;
        thread_kvp.value     = max_sum;
        const kvp result_kvp = multithread_reduce(thread_kvp, arg_min, params.THREADS_PER_ROW);
        expert               = result_kvp.key;

        // clear the max value in the thread
        if(k_idx < params.THREADS_PER_ROW - topk_group)
        {
            int const thread_to_clear_in_group = expert / params.VPT;

            if(thread_group_idx == thread_to_clear_in_group)
            {
                bias_chunk[0] = FLT_MAX;
                max_val       = FLT_MAX;
            }
        }
    }

    // __syncthreads();

    ////////////////////// Topk //////////////////////
    float output_sum = 0.0f;
    // uint32_t expert_mask = 0xFFFFFFFF;
    for(int k_idx = 0; k_idx < topk_excluding_share_expert_fusion; ++k_idx)
    {
        // local argmax
        float max_val = bias_chunk[0];
        int expert    = first_elt_read_by_thread;

        if(!cmp_eq(max_val, FLT_MAX))
        {
#pragma unroll
            for(int ii = 1; ii < params.VPT; ++ii)
            {
                float val = bias_chunk[ii];
                // if (((expert_mask >> ii) & 1u) && cmp_gt(val, max_val)) {
                if(cmp_gt(val, max_val))
                {
                    max_val = val;
                    expert  = first_elt_read_by_thread + ii;
                }
            }
        }
        else
        {
            max_val = -FLT_MAX;
        }

        using kvp = hipcub::KeyValuePair<int, float>;
        hipcub::ArgMax arg_max;
        kvp thread_kvp;
        thread_kvp.key       = expert;
        thread_kvp.value     = max_val;
        const kvp result_kvp = multithread_reduce(thread_kvp, arg_max, params.THREADS_PER_ROW);
        expert               = result_kvp.key;

        //     // argmax reduce
        // #pragma unroll
        //     for (int mask = params.THREADS_PER_ROW / 2; mask > 0; mask /= 2) {
        //       float other_max =
        //           VLLM_SHFL_XOR_SYNC_WIDTH(max_val, mask, params.THREADS_PER_ROW);
        //       int other_expert = VLLM_SHFL_XOR_SYNC_WIDTH(expert, mask, params.THREADS_PER_ROW);
        //       // float other_scale = VLLM_SHFL_XOR_SYNC_WIDTH(scale, mask,
        //       params.THREADS_PER_ROW);

        //       // lower indices to win
        //       if (cmp_gt(other_max, max_val) || (cmp_eq(other_max, max_val) && other_expert <
        //       expert)) {
        //         max_val = other_max;
        //         expert = other_expert;
        //         // scale = other_scale;
        //       }
        //     }

        int thread_to_clear_in_group = expert / params.VPT;
        int64_t idx                  = topk * thread_row + k_idx;

        if(thread_group_idx == thread_to_clear_in_group)
        {
            int expert_to_clear_in_thread = expert % params.VPT;
            // topk_indices[k_idx] = expert;

#pragma unroll
            for(int ii = 0; ii < params.VPT; ++ii)
            {
                if(ii == expert_to_clear_in_thread)
                {
                    bias_chunk[ii] = -FLT_MAX; // clear the max value in the thread
                    // output_ptr[idx] = row_chunk[ii];
                    scores[k_idx] = row_chunk[ii];
                }
            }
            // output_ptr[idx] = row_chunk[k_idx];
            // expert_mask &= ~(1u << expert_to_clear_in_thread);
            // output_ptr[idx] = scale;  // store output

            //// clear the max value in the thread
            // bias_chunk[expert_to_clear_in_thread] = -FLT_MAX;
            //// store output
            // output_ptr[idx] = row_chunk[expert_to_clear_in_thread];
            indices_ptr[idx] = ck_tile::type_convert<int32_t>(expert);
        }
        __syncthreads();

        // accumulate sum for all elements
        if(thread_group_idx == 0)
        {
            // output_sum += output_ptr[idx];
            output_sum += scores[k_idx];
        }

        // __syncthreads();
    }

    if(thread_group_idx == 0 && num_fused_shared_experts > 0)
    {
        int64_t last_idx = topk * thread_row + topk_excluding_share_expert_fusion;

        // Use round-robin to select expert
        int64_t expert_offset = thread_row % num_fused_shared_experts;
        indices_ptr[last_idx] = ck_tile::type_convert<int32_t>(params.NUM_EXPERTS + expert_offset);

        // Set the weight to the sum of all weights divided by routed_scaling_factor
        output_ptr[last_idx] = output_sum / routed_scaling_factor;

        if(num_fused_shared_experts > 1)
        {
            for(int i = 1; i < num_fused_shared_experts; ++i)
            {
                ++last_idx;
                ++expert_offset;
                indices_ptr[last_idx] = static_cast<int32_t>(params.NUM_EXPERTS + expert_offset);
                // Set the weight to the sum of all weights divided by routed_scaling_factor
                output_ptr[last_idx] = output_sum / routed_scaling_factor;
            }
        }
    }
    __syncthreads();

    ////////////////////// Rescale Output //////////////////////
    if(thread_group_idx == 0)
    {
#pragma unroll
        for(int ii = 0; ii < topk; ++ii)
        {
            int64_t const idx = topk * thread_row + ii;
            output_ptr[idx]   = scores[ii] / output_sum;
        }
    }
}

//------------------------------------------------------------------------------
// Templated Kernel Version (using compile-time constants)
//------------------------------------------------------------------------------
template <int VPT_,
          int NUM_EXPERTS_,
          int THREADS_PER_ROW_,
          int ROWS_PER_WARP_,
          int ROWS_PER_CTA_,
          int WARPS_PER_CTA_>
struct KernelParams
{
    static constexpr int VPT             = VPT_;
    static constexpr int NUM_EXPERTS     = NUM_EXPERTS_;
    static constexpr int THREADS_PER_ROW = THREADS_PER_ROW_;
    static constexpr int ROWS_PER_WARP   = ROWS_PER_WARP_;
    static constexpr int ROWS_PER_CTA    = ROWS_PER_CTA_;
    static constexpr int WARPS_PER_CTA   = WARPS_PER_CTA_;
};

template <typename T,
          int VPT,
          int NUM_EXPERTS,
          int THREADS_PER_ROW,
          int ROWS_PER_WARP,
          int ROWS_PER_CTA,
          int WARPS_PER_CTA>
__global__ void moe_fused_gate_kernel(void* input,
                                      void* bias,
                                      float* output_ptr,
                                      int32_t* indices_ptr,
                                      int64_t num_rows,
                                      int64_t topk_group,
                                      int64_t topk,
                                      int64_t num_fused_shared_experts,
                                      double routed_scaling_factor)
{
    KernelParams<VPT, NUM_EXPERTS, THREADS_PER_ROW, ROWS_PER_WARP, ROWS_PER_CTA, WARPS_PER_CTA>
        params;
    moe_fused_gate_impl<T>(input,
                           bias,
                           output_ptr,
                           indices_ptr,
                           num_rows,
                           topk_group,
                           topk,
                           num_fused_shared_experts,
                           routed_scaling_factor,
                           params);
}

// Macro to compute compile-time constants and launch the kernel.
#define LAUNCH_MOE_GATE_CONFIG(T, EXPERTS, EXPERT_GROUP)                                      \
    do                                                                                        \
    {                                                                                         \
        constexpr int VPT = (EXPERTS) / (EXPERT_GROUP);                                       \
        /* If EXPERT_GROUP > WARP_SIZE, fall back to 1 row per warp */                        \
        constexpr int ROWS_PER_WARP =                                                         \
            ((EXPERT_GROUP) <= WARP_SIZE) ? (WARP_SIZE / (EXPERT_GROUP)) : 1;                 \
        constexpr int ROWS_PER_CTA = WARPS_PER_CTA * ROWS_PER_WARP;                           \
        moe_fused_gate_kernel<T,                                                              \
                              VPT,                                                            \
                              (EXPERTS),                                                      \
                              (EXPERT_GROUP),                                                 \
                              ROWS_PER_WARP,                                                  \
                              ROWS_PER_CTA,                                                   \
                              WARPS_PER_CTA>                                                  \
            <<<num_blocks, block_dim, shared_mem_size, stream>>>(input.data_ptr(),            \
                                                                 bias.data_ptr(),             \
                                                                 output.data_ptr<float>(),    \
                                                                 indices.data_ptr<int32_t>(), \
                                                                 num_rows,                    \
                                                                 topk_group,                  \
                                                                 topk,                        \
                                                                 num_fused_shared_experts,    \
                                                                 routed_scaling_factor);      \
        dispatched = true;                                                                    \
    } while(0)

//------------------------------------------------------------------------------
// Dynamic Kernel Version (parameters computed at runtime)
//------------------------------------------------------------------------------
struct KernelParamsDynamic
{
    int VPT;
    int NUM_EXPERTS;
    int THREADS_PER_ROW;
    int ROWS_PER_WARP;
    int ROWS_PER_CTA;
    int WARPS_PER_CTA;
};

template <typename T>
__global__ void moe_fused_gate_kernel_dynamic(void* input,
                                              void* bias,
                                              float* output_ptr,
                                              int32_t* indices_ptr,
                                              int64_t num_rows,
                                              int64_t num_experts,
                                              int64_t num_expert_group,
                                              int64_t topk_group,
                                              int64_t topk,
                                              int64_t num_fused_shared_experts,
                                              double routed_scaling_factor)
{
    KernelParamsDynamic params;
    params.NUM_EXPERTS = num_experts;            // e.g, for deepseek v3, this is 256
    params.VPT = num_experts / num_expert_group; // e.g., for deepseek v3, this is 256 / 8 = 32
    params.THREADS_PER_ROW =
        num_expert_group; // fixed as num_expert_group, e.g., for deepseek v3, this is 8
    params.WARPS_PER_CTA = WARPS_PER_CTA; // fixed as 6
    params.ROWS_PER_WARP =
        std::max<int64_t>(1, WARP_SIZE / num_expert_group); // WARP_SIZE is fixed as 32
    params.ROWS_PER_CTA = params.WARPS_PER_CTA * params.ROWS_PER_WARP;

    moe_fused_gate_impl<T>(input,
                           bias,
                           output_ptr,
                           indices_ptr,
                           num_rows,
                           topk_group,
                           topk,
                           num_fused_shared_experts,
                           routed_scaling_factor,
                           params);
}

//------------------------------------------------------------------------------
// Host Launcher Function
//------------------------------------------------------------------------------
std::vector<at::Tensor> moe_fused_gate(at::Tensor& input,
                                       at::Tensor& bias,
                                       at::Tensor& topk_weights,
                                       at::Tensor& topk_ids,
                                       int64_t num_expert_group,
                                       int64_t topk_group,
                                       int64_t topk,
                                       int64_t num_fused_shared_experts,
                                       double routed_scaling_factor)
{
    int64_t num_rows    = input.size(0);
    int32_t num_experts = input.size(1);
    auto options        = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
    auto output         = topk_weights;
    auto indices        = topk_ids;

    // Compute grid dimensions based on runtime value for num_expert_group.
    int64_t rows_per_warp = std::max<int64_t>(1, WARP_SIZE / num_expert_group);
    int64_t num_warps     = (num_rows + rows_per_warp - 1) / rows_per_warp;
    int64_t num_blocks    = (num_warps + WARPS_PER_CTA - 1) / WARPS_PER_CTA;
    int ROWS_PER_WARP     = std::max<int64_t>(1, WARP_SIZE / num_expert_group);
    size_t shared_mem_size =
        ((topk * sizeof(float) + topk * sizeof(int)) * ROWS_PER_WARP + 255) & ~255;
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    dim3 block_dim(WARP_SIZE, WARPS_PER_CTA);

    // Check 1: Ensure that num_experts is a power of 2.
    TORCH_CHECK((num_experts & (num_experts - 1)) == 0,
                "num_experts must be a power of 2, but got ",
                num_experts);

    // Check 2: Ensure that num_experts is divisible by num_expert_group. (this also means
    // num_expert_group is power of 2)
    TORCH_CHECK(num_experts % num_expert_group == 0,
                "num_experts must be divisible by num_expert_group, but got ",
                num_experts,
                " / ",
                num_expert_group);

    int computed_vpt = num_experts / num_expert_group;
    // Check 3: Ensure that num_experts/num_expert_group does not exceed MAX_VPT=32. Maximum VPT
    // indicate max value per threads we can process.
    TORCH_CHECK(computed_vpt <= MAX_VPT,
                "Per group experts: num_experts / num_expert_group = (",
                computed_vpt,
                ") exceeds the maximum supported (",
                MAX_VPT,
                ")");

    // Dispatch to templated kernel for known compile-time configurations.
    // We currently only support for:
    //   Case 1: 256 experts, with 8 or 16 groups.
    //   Case 2: 128 experts, with 4 or 8 groups.
    //   Case 3: other cases, require 8 <= num_experts / num_expert_group <= 32
    bool dispatched = false;
    switch(num_experts)
    {
    case 256:
        if(num_expert_group == 8)
            // This is deepseek v3 case. Here VPT = 256/8 = 32, ROWS_PER_WARP = 32/8 = 4,
            // ROWS_PER_CTA = 6 * 4 = 24.
            if(input.scalar_type() == at::kBFloat16)
            {
                LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 256, 8);
            }
            else if(input.scalar_type() == at::kHalf)
            {
                LAUNCH_MOE_GATE_CONFIG(float16_t, 256, 8);
            }
            else if(input.scalar_type() == at::kFloat)
            {
                LAUNCH_MOE_GATE_CONFIG(float32_t, 256, 8);
            }
            else if(num_expert_group == 16)
                // Here VPT = 256/16 = 16, ROWS_PER_WARP = 32/16 = 2, ROWS_PER_CTA = 6 * 2 = 12.
                if(input.scalar_type() == at::kBFloat16)
                {
                    LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 256, 16);
                }
                else if(input.scalar_type() == at::kHalf)
                {
                    LAUNCH_MOE_GATE_CONFIG(float16_t, 256, 16);
                }
                else if(input.scalar_type() == at::kFloat)
                {
                    LAUNCH_MOE_GATE_CONFIG(float32_t, 256, 16);
                }
        break;
    case 128:
        if(num_expert_group == 4)
            // VPT = 128/4 = 32, ROWS_PER_WARP = 32/16 = 2, ROWS_PER_CTA = 6 * 2 = 12.
            if(input.scalar_type() == at::kBFloat16)
            {
                LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 128, 4);
            }
            else if(input.scalar_type() == at::kHalf)
            {
                LAUNCH_MOE_GATE_CONFIG(float16_t, 128, 4);
            }
            else if(input.scalar_type() == at::kFloat)
            {
                LAUNCH_MOE_GATE_CONFIG(float32_t, 128, 4);
            }
            else if(num_expert_group == 8)
                // VPT = 128/8 = 16, ROWS_PER_WARP = 32/8 = 4, ROWS_PER_CTA = 6 * 4 = 24.
                if(input.scalar_type() == at::kBFloat16)
                {
                    LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 128, 8);
                }
                else if(input.scalar_type() == at::kHalf)
                {
                    LAUNCH_MOE_GATE_CONFIG(float16_t, 128, 8);
                }
                else if(input.scalar_type() == at::kFloat)
                {
                    LAUNCH_MOE_GATE_CONFIG(float32_t, 128, 8);
                }
        break;
    default: break;
    }
    if(!dispatched)
    {
        // Fallback to the dynamic kernel if none of the supported combinations match.
        // currently only support num_experts / num_expert_group <= 32 for dynamic kernels
        if(input.scalar_type() == at::kBFloat16)
        {
            moe_fused_gate_kernel_dynamic<bfloat16_t>
                <<<num_blocks, block_dim, shared_mem_size, stream>>>(input.data_ptr(),
                                                                     bias.data_ptr(),
                                                                     output.data_ptr<float>(),
                                                                     indices.data_ptr<int32_t>(),
                                                                     num_rows,
                                                                     num_experts,
                                                                     num_expert_group,
                                                                     topk_group,
                                                                     topk,
                                                                     num_fused_shared_experts,
                                                                     routed_scaling_factor);
        }
        else if(input.scalar_type() == at::kHalf)
        {
            moe_fused_gate_kernel_dynamic<float16_t>
                <<<num_blocks, block_dim, shared_mem_size, stream>>>(input.data_ptr(),
                                                                     bias.data_ptr(),
                                                                     output.data_ptr<float>(),
                                                                     indices.data_ptr<int32_t>(),
                                                                     num_rows,
                                                                     num_experts,
                                                                     num_expert_group,
                                                                     topk_group,
                                                                     topk,
                                                                     num_fused_shared_experts,
                                                                     routed_scaling_factor);
        }
        else if(input.scalar_type() == at::kFloat)
        {
            moe_fused_gate_kernel_dynamic<float32_t>
                <<<num_blocks, block_dim, shared_mem_size, stream>>>(input.data_ptr(),
                                                                     bias.data_ptr(),
                                                                     output.data_ptr<float>(),
                                                                     indices.data_ptr<int32_t>(),
                                                                     num_rows,
                                                                     num_experts,
                                                                     num_expert_group,
                                                                     topk_group,
                                                                     topk,
                                                                     num_fused_shared_experts,
                                                                     routed_scaling_factor);
        }
        else
        {
            TORCH_CHECK(false, "Unsupported data type for moe_fused_gate");
        }
    }
    return {output, indices};
}
